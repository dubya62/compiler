#include "hip/hip_runtime.h"

__global__ void add(int* a, int* b, int* c){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
}

int main(int argc, char** argv){
    int* a = {1, 2, 3};
    int* b = {1, 2, 3};
    int* c;
    add<<<1, 3>>>(a, b, c);
    return 0;
}
